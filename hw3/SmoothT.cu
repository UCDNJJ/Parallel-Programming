#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>
#include <cstdlib>
#include <algorithm>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/tuple.h>
#include <thrust/sequence.h>

/*#define STANDALONE*/
/*#define TEST*/
/*#define PRINT*/

#ifdef STANDALONE
#include <sys/time.h>

#define MICRO_IN_SEC 1000000

typedef unsigned long long timestamp_t;

timestamp_t get_timestamp()
{
    struct timeval now;
    gettimeofday(&now, NULL);
    return now.tv_usec + (timestamp_t)(now.tv_sec) * MICRO_IN_SEC;
}

void checkpoint(timestamp_t* marker, const char* message)
{
    timestamp_t now = get_timestamp();
    printf("# %20f - %s\n", (now-*marker) / (double)MICRO_IN_SEC, message);
    *marker = now;
}
#endif

struct SumAndCount {
    float *j_x,
          *j_y;
    int j_n;
    float h;

    SumAndCount(thrust::device_vector<float>::iterator it_x,
            thrust::device_vector<float>::iterator it_y,
            int j_n, float h) : j_n(j_n), h(h) {
        j_x = thrust::raw_pointer_cast(&it_x[0]);
        j_y = thrust::raw_pointer_cast(&it_y[0]);
    }

    __device__ thrust::tuple<float, int> operator()(
            thrust::tuple<float, float, int> ix_sum_count) {
        float ix = thrust::get<0>(ix_sum_count);
        float sum = thrust::get<1>(ix_sum_count);
        int count = thrust::get<2>(ix_sum_count);

        for(int j = 0; j < j_n; j++) {
            if(abs(j_x[j] - ix) < h) {
                sum += j_y[j];
                count += 1;
            }
        }

        return thrust::make_tuple(sum, count);
    }
};

struct Mean {
    __device__ float operator()(const float sum, const float count) {
        return sum / count;
    }
};

void smootht(float *x, float *y, float *m, int n, float h) {
    thrust::host_vector<float> host_x(x, x + n);
    thrust::host_vector<float> host_y(y, y + n);
    thrust::host_vector<float> host_m(m, m + n);
    thrust::host_vector<int> host_count(n);
    thrust::fill(host_count.begin(), host_count.end(), 0);
    thrust::fill(host_m.begin(), host_m.end(), 0);

    int chunk_size = 100000;
    int chunk_count = ceil(n / (double)chunk_size);

    thrust::device_vector<float> device_i_x(chunk_size);
    thrust::device_vector<float> device_j_x(chunk_size);
    thrust::device_vector<float> device_j_y(chunk_size);
    thrust::device_vector<int> device_i_count(chunk_size);
    thrust::device_vector<float> device_i_m(chunk_size);
    thrust::device_vector<int> device_out_count(chunk_size);
    thrust::device_vector<float> device_out_m(chunk_size);
    thrust::device_vector<int> seq(chunk_size);
    thrust::sequence(seq.begin(), seq.end());

    for(int i = 0; i < chunk_count; i++) {
        for(int j = 0; j < chunk_count; j++) {
            int i_remaining_count = std::min(chunk_size, n - i*chunk_size);
            int j_remaining_count = std::min(chunk_size, n - j*chunk_size);

            thrust::copy(host_x.begin() + i*chunk_size,
                    host_x.begin() + i*chunk_size + i_remaining_count,
                    device_i_x.begin());

            thrust::copy(host_x.begin() + j*chunk_size,
                    host_x.begin() + j*chunk_size + j_remaining_count,
                    device_j_x.begin());
            thrust::copy(host_y.begin() + j*chunk_size,
                    host_y.begin() + j*chunk_size + j_remaining_count,
                    device_j_y.begin());

            thrust::copy(host_m.begin() + i*chunk_size,
                    host_m.begin() + i*chunk_size + i_remaining_count,
                    device_i_m.begin());
            thrust::copy(host_count.begin() + i*chunk_size,
                    host_count.begin() + i*chunk_size + i_remaining_count,
                    device_i_count.begin());

            thrust::transform(
                    thrust::make_zip_iterator(thrust::make_tuple(
                            device_i_x.begin(),
                            device_i_m.begin(),
                            device_i_count.begin())),
                    thrust::make_zip_iterator(thrust::make_tuple(
                            device_i_x.begin() + i_remaining_count,
                            device_i_m.begin() + i_remaining_count,
                            device_i_count.begin() + i_remaining_count)),
                    thrust::make_zip_iterator(thrust::make_tuple(
                            device_out_m.begin(), device_out_count.begin())),
                    SumAndCount(device_j_x.begin(), device_j_y.begin(),
                        j_remaining_count, h));

            thrust::copy(device_out_m.begin(),
                    device_out_m.begin() + i_remaining_count,
                    host_m.begin() + i*chunk_size);
            thrust::copy(device_out_count.begin(),
                    device_out_count.begin() + i_remaining_count,
                    host_count.begin() + i*chunk_size);
        }
    }

    for(int i = 0; i < chunk_count; i++) {
        int i_remaining_count = std::min(chunk_size, n - i*chunk_size);

        thrust::copy(host_m.begin() + i*chunk_size,
                host_m.begin() + i*chunk_size + i_remaining_count,
                device_i_m.begin());
        thrust::copy(host_count.begin() + i*chunk_size,
                host_count.begin() + i*chunk_size + i_remaining_count,
                device_i_count.begin());

        thrust::transform(device_i_m.begin(),
                device_i_m.begin() + i_remaining_count,
                device_i_count.begin(),
                device_out_m.begin(),
                Mean());

        thrust::copy(device_out_m.begin(),
                device_out_m.begin() + i_remaining_count,
                host_m.begin() + i*chunk_size);
    }

    thrust::copy(host_m.begin(), host_m.end(), m);
#ifdef PRINT
    for(int i = 0; i < 10; i++) {
        printf("%f\n", m[i]);
    }
#endif
}

#ifdef STANDALONE
int main(int argc, char **argv) {
    int element_count;
    int h;
#ifdef TEST
    h = 2;
    element_count = 10;
#else
    h = 100000;
    FILE* fp = fopen("test.txt", "r");
    fscanf(fp, "%d\n", &element_count);
    element_count = 100000;
#endif
    int float_byte_count = element_count * sizeof(float);
    float *host_x = (float*)malloc(float_byte_count);
    float *host_y = (float*)malloc(float_byte_count);
    float *host_m = (float*)malloc(float_byte_count);
#ifdef TEST
    for(int i = 0; i < element_count; i++) {
        host_x[i] = i;
        host_y[i] = i;
    }
#else
    for(int i = 0; i < element_count; i++) {
        fscanf(fp, "%f\n", host_x+i);
        host_y[i] = host_x[i] + host_x[0];
    }
#endif
    timestamp_t marker = get_timestamp();
    smootht(host_x, host_y, host_m, element_count, h);
    checkpoint(&marker, "TOTAL");
#ifdef PRINT
    for(int i = 0; i < element_count; i++) {
        printf("(%f, %f, %f)\n", host_x[i], host_y[i], host_m[i]);
    }
#endif
    return 0;
}
#endif
