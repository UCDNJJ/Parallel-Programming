#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>
#include <algorithm>

/*#define STANDALONE*/
/*#define TEST*/
/*#define PRINT*/

#ifdef STANDALONE
#include <sys/time.h>

#define MICRO_IN_SEC 1000000

typedef unsigned long long timestamp_t;

timestamp_t get_timestamp()
{
    struct timeval now;
    gettimeofday(&now, NULL);
    return now.tv_usec + (timestamp_t)(now.tv_sec) * MICRO_IN_SEC;
}

void checkpoint(timestamp_t* marker, const char* message)
{
    timestamp_t now = get_timestamp();
    printf("# %20f - %s\n", (now-*marker) / (double)MICRO_IN_SEC, message);
    *marker = now;
}
#endif

__global__ void sum_and_count(float *i_x, int i_n, float *j_x, float *j_y, int j_n,
        float h, float *i_sum, int *i_count) {
    int me_i = blockIdx.x * blockDim.x + threadIdx.x;

    float my_sum = 0;
    int my_count = 0;
    
    if(me_i < i_n) {
        for(int j = 0; j < j_n; j++) {
            if(abs(j_x[j] - i_x[me_i]) < h) {
#ifdef PRINT
                /*printf("%d, %d, value: %f\n", me_i, j, j_y[j]);*/
#endif
                my_sum += j_y[j];
                my_count += 1;
            }
        }
    }

    i_sum[me_i] = my_sum;
    i_count[me_i] = my_count;
}

__global__ void div(float *m, int *count, int n) {
    int me = blockIdx.x * blockDim.x + threadIdx.x;
    if(me < n) {
#ifdef PRINT
        /*printf("mean: %f, %d\n", m[me], count[me]);*/
#endif
        m[me] /= count[me];
    }
}

#define CUDA_CHECK(EXP) if(EXP != hipSuccess) goto hipError_t;

void smoothc(float *x, float *y, float *m, int n, float h) {
    float *device_i_x = NULL,
        *device_i_y = NULL,
        *device_j_x = NULL,
        *device_j_y = NULL,
        *device_i_m = NULL;

    int *host_count,
        *device_i_count;

    int int_byte_count = n * sizeof(int);

    host_count = (int*)malloc(int_byte_count);

    for(int i = 0; i < n; i++) {
        m[i] = 0;
        host_count[i] = 0;
    }

    int chunk_size = 100000;
    int chunk_float_byte_count = chunk_size * sizeof(float);
    int chunk_int_byte_count = chunk_size * sizeof(int);
    int chunk_count = ceil(n / (double)chunk_size);
#ifdef PRINT
    printf("%d\n", chunk_count);
#endif

    int dim_block = 192;
    int dim_grid = ceil(chunk_size / (double)dim_block);
#ifdef PRINT
    printf("Conf: %d %d\n", dim_block, dim_grid);
#endif

    CUDA_CHECK(hipMalloc((void **)&device_i_x,     chunk_float_byte_count));
    CUDA_CHECK(hipMalloc((void **)&device_j_x,     chunk_float_byte_count));
    CUDA_CHECK(hipMalloc((void **)&device_j_y,     chunk_float_byte_count));
    CUDA_CHECK(hipMalloc((void **)&device_i_count, chunk_int_byte_count));
    CUDA_CHECK(hipMalloc((void **)&device_i_m,     chunk_float_byte_count));

    for(int i = 0; i < chunk_count; i++) {
        for(int j = 0; j < chunk_count; j++) {
            int i_remaining_count = std::min(chunk_size, n - i*chunk_size);
            int j_remaining_count = std::min(chunk_size, n - j*chunk_size);

            CUDA_CHECK(hipMemcpy(device_i_x, x + i*chunk_size,
                        i_remaining_count*sizeof(float), hipMemcpyHostToDevice));

            CUDA_CHECK(hipMemcpy(device_j_x, x + j*chunk_size,
                        j_remaining_count*sizeof(float), hipMemcpyHostToDevice));
            CUDA_CHECK(hipMemcpy(device_j_y, y + j*chunk_size,
                        j_remaining_count*sizeof(float), hipMemcpyHostToDevice));

            CUDA_CHECK(hipMemcpy(device_i_m, m + i*chunk_size,
                        i_remaining_count*sizeof(float), hipMemcpyHostToDevice));
            CUDA_CHECK(hipMemcpy(device_i_count, host_count + i*chunk_size,
                        i_remaining_count*sizeof(int), hipMemcpyHostToDevice));

            sum_and_count<<<dim_grid, dim_block>>>(device_i_x, 
                    i_remaining_count,
                    device_j_x, device_j_y,
                    j_remaining_count,
                    h, device_i_m, device_i_count);

            CUDA_CHECK(hipMemcpy(m + i*chunk_size, device_i_m,
                        i_remaining_count*sizeof(float), hipMemcpyDeviceToHost));
            CUDA_CHECK(hipMemcpy(host_count + i*chunk_size, device_i_count,
                        i_remaining_count*sizeof(float), hipMemcpyDeviceToHost));
        }
    }
    for(int i = 0; i < chunk_count; i++) {
        int i_remaining_count = std::min(chunk_size, n - i*chunk_size);
        CUDA_CHECK(hipMemcpy(device_i_m, m + i*chunk_size,
                    i_remaining_count*sizeof(float), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(device_i_count, host_count + i*chunk_size,
                    i_remaining_count*sizeof(int), hipMemcpyHostToDevice));

        div<<<dim_grid, dim_block>>>(device_i_m, device_i_count,
                i_remaining_count);
        
        CUDA_CHECK(hipMemcpy(m + i*chunk_size, device_i_m,
                    i_remaining_count*sizeof(float), hipMemcpyDeviceToHost));
    }

hipError_t:
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(error));
    }
    free(host_count);
    hipFree(device_i_x);
    hipFree(device_i_y);
    hipFree(device_i_count);
    hipFree(device_i_m);
}

#ifdef STANDALONE
int main(int argc, char **argv) {
    int element_count;
    int h;
#ifdef TEST
    h = 2;
    element_count = 10;
#else
    h = 100000;
    FILE* fp = fopen("test.txt", "r");
    fscanf(fp, "%d\n", &element_count);
    element_count = 100000;
#endif
    int float_byte_count = element_count * sizeof(float);
    float *host_x = (float*)malloc(float_byte_count);
    float *host_y = (float*)malloc(float_byte_count);
    float *host_m = (float*)malloc(float_byte_count);
#ifdef TEST
    for(int i = 0; i < element_count; i++) {
        host_x[i] = i;
        host_y[i] = i;
    }
#else
    for(int i = 0; i < element_count; i++) {
        fscanf(fp, "%f\n", host_x+i);
        host_y[i] = host_x[i] + host_x[0];
    }
#endif
    timestamp_t marker = get_timestamp();
    smoothc(host_x, host_y, host_m, element_count, h);
    checkpoint(&marker, "TOTAL");
#ifdef PRINT
    for(int i = 0; i < element_count; i++) {
        printf("(%f, %f, %f)\n", host_x[i], host_y[i], host_m[i]);
    }
#endif
    return 0;
}
#endif
